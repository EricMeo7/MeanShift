#include "hip/hip_runtime.h"
#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <chrono>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

#include "hip/hip_vector_types.h"
#include ""

#include <vector>
#include <iostream>
#include <fstream>
#include <sstream>

#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)
#define BANDWIDTH 2
#define NUM_DIMENSIONS 3
#define NUM_ITERATIONS 10
#define TILE_WIDTH 64
#define BLOCK_DIM TILE_WIDTH


__global__ void MeanShiftWithoutTiling(float* shiftedPoints, const float* __restrict__ originalPoints, const unsigned numPoints) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float3 newPosition = make_float3(0.0, 0.0, 0.0);
	float totalWeight = 0.0;

	if (idx < numPoints) {
		float x = shiftedPoints[idx];
		float y = shiftedPoints[idx + numPoints];
		float z = shiftedPoints[idx + 2 * numPoints];
		float3 shiftedPoint = make_float3(x, y, z);

		for (int i = 0; i < numPoints; i++) {
			x = originalPoints[i];
			y = originalPoints[i + numPoints];
			z = originalPoints[i + 2 * numPoints];
			float3 originalPoint = make_float3(x, y, z);
			float3 difference = shiftedPoint - originalPoint;
			float squaredDistance = dot(difference, difference);
			float weight = std::exp((-squaredDistance) / (2 * powf(BANDWIDTH, 2)));
			newPosition += originalPoint * weight;
			totalWeight += weight;
		}
		newPosition /= totalWeight;
		shiftedPoints[idx] = newPosition.x;
		shiftedPoints[idx + numPoints] = newPosition.y;
		shiftedPoints[idx + 2 * numPoints] = newPosition.z;
	}
}

__global__ void MeanShiftTiling(float* shiftedPoints, const float* __restrict__ originalPoints, const unsigned numPoints) {

	__shared__ float tile[TILE_WIDTH][3];

	int tx = threadIdx.x;
	int idx = blockIdx.x * blockDim.x + tx;

	float3 newPosition = make_float3(0.0, 0.0, 0.0);
	float totalWeight = 0.0;
	
	// loading phase - each thread load something into shared memory
	for (int tile_i = 0; tile_i < (numPoints - 1) / TILE_WIDTH + 1; ++tile_i) {

		int tile_idx = tile_i * TILE_WIDTH + tx;

		if (tile_idx < numPoints) {
			tile[tx][0] = originalPoints[tile_idx];
			tile[tx][1] = originalPoints[tile_idx + numPoints];
			tile[tx][2] = originalPoints[tile_idx + 2 * numPoints];
		}
		else {
			tile[tx][0] = 0.0;
			tile[tx][1] = 0.0;
			tile[tx][2] = 0.0;
		}

		__syncthreads();
	}
	//end of loading into shared memory
	
	//computing phase
	// only the threads inside the bounds do some computation
	if (idx < numPoints) {
		float x = shiftedPoints[idx];
		float y = shiftedPoints[idx + numPoints];
		float z = shiftedPoints[idx + 2 * numPoints];
		float3 shiftedPoint = make_float3(x, y, z);

		for (int i = 0; i < TILE_WIDTH; i++) {
			if (tile[i][0] != 0.0 && tile[i][1] != 0.0 && tile[i][2] != 0.0) {
				float3 originalPoint = make_float3(tile[i][0], tile[i][1], tile[i][2]);
				float3 difference = shiftedPoint - originalPoint;
				float squaredDistance = dot(difference, difference);
				if (sqrt(squaredDistance) <= BANDWIDTH) {
					float weight = std::exp((-squaredDistance) / (2 * powf(BANDWIDTH, 2)));
					newPosition += originalPoint * weight;
					totalWeight += weight;
				}
			}
		}
	}
	__syncthreads();
	
	if (idx < numPoints) {
		newPosition /= totalWeight;
		shiftedPoints[idx] = newPosition.x;
		shiftedPoints[idx + numPoints] = newPosition.y;
		shiftedPoints[idx + 2 * numPoints] = newPosition.z;
	}

}

int main(void)
{
	/* inserico i punti dal csv su vettore points - inzio */
 	float time;
	//std::string fileName = "dataset/3D_data_100.csv";
	//std::string fileName = "dataset/3D_data_1000.csv";
	//std::string fileName = "dataset/3D_data_10000.csv";
	//std::string fileName = "dataset/3D_data_20000.csv";
	//std::string fileName = "dataset/3D_data_100000.csv";
	//std::string fileName = "dataset/3D_data_250000.csv";
	//std::string fileName = "dataset/3D_data_500000.csv";
	//std::string fileName = "dataset/3D_data_1000000.csv";
	std::string fileName = "dataset/3D_data_2000000.csv";
	
	std::chrono::high_resolution_clock::time_point start;
	std::chrono::high_resolution_clock::time_point end;
	std::vector<float> h_inputPoints;
	std::vector<float> x;
	std::vector<float> y;
	std::vector<float> z;
	std::ifstream data(fileName);
	std::string line;
	while (std::getline(data, line)) {
		std::vector<float> point;
		std::stringstream lineStream(line);
		std::string cell;
		while (std::getline(lineStream, cell, ',')) {
			point.push_back(stod(cell));
		}
		x.push_back(point[0]);
		y.push_back(point[1]);
		z.push_back(point[2]);
	}
	h_inputPoints = x;
	h_inputPoints.insert(h_inputPoints.end(), y.begin(), y.end());
	h_inputPoints.insert(h_inputPoints.end(), z.begin(), z.end());
	/* inserico i punti dal csv su vettore points - fine */

	int numPoints = h_inputPoints.size() / NUM_DIMENSIONS;
	printf("Numero di punti %d\n", numPoints);


	dim3 gridDim = dim3(ceil((float)numPoints / BLOCK_DIM));
	dim3 blockDim = dim3(BLOCK_DIM);

	/* TEST CUDA CON TILING - inizio */
	 

	/* Copy host_vector to device_vector */
	thrust::device_vector<float> d_originalPoints2 = h_inputPoints;
	thrust::device_vector<float> d_shiftedPoints2 = h_inputPoints;

	start = std::chrono::high_resolution_clock::now();
	for (int i = 0; i < NUM_ITERATIONS; i++) {
		MeanShiftTiling<<< gridDim, blockDim >>> (thrust::raw_pointer_cast(&d_shiftedPoints2[0]), thrust::raw_pointer_cast(&d_originalPoints2[0]), numPoints);
		hipDeviceSynchronize();
	}

	end = std::chrono::high_resolution_clock::now();
	time = std::chrono::duration_cast<std::chrono::duration<float>>(end - start).count();
	 

	printf("\nTempo esecuzione Mean Shift con tiling: %f\n", time);
	/* TEST CUDA CON TILING - fine */
	/*d_originalPoints.clear();
	d_originalPoints.shrink_to_fit();
	d_shiftedPoints.clear();
	d_shiftedPoints.shrink_to_fit();*/
	
	/* TEST CUDA SENZA TILING - inizio*/
	 
	
	/* Copy host_vector to device_vector */
	thrust::device_vector<float> d_originalPoints = h_inputPoints;
	thrust::device_vector<float> d_shiftedPoints = h_inputPoints;
	start = std::chrono::high_resolution_clock::now();
	for (int i = 0; i < NUM_ITERATIONS; i++) {
		MeanShiftWithoutTiling <<<gridDim, blockDim>>> (thrust::raw_pointer_cast(&d_shiftedPoints[0]), thrust::raw_pointer_cast(&d_originalPoints[0]), numPoints);
		hipDeviceSynchronize();
	}

	end = std::chrono::high_resolution_clock::now();
	time = std::chrono::duration_cast<std::chrono::duration<float>>(end - start).count();
	

	printf("\nTempo esecuzione Mean Shift senza tiling: %f\n", time);
	/* TEST CUDA SENZA TILING  - fine */


	/* CLUSTERING - inizio */
	/* Copy device_vector to host_vector */
	thrust::host_vector<float> h_ShiftedPoints = d_shiftedPoints;

	std::vector<float> clusterPoints;
	clusterPoints.resize(numPoints);

	start = std::chrono::high_resolution_clock::now();
	std::vector<float3> clusters;
	float clusterEps = 5;
	for (int i = 0; i < numPoints; i++) {
		float x = h_ShiftedPoints[i];
		float y = h_ShiftedPoints[i + numPoints];
		float z = h_ShiftedPoints[i + 2 * numPoints];
		float3 point = make_float3(x, y, z);
		auto iter = clusters.begin();
		auto iterEnd = clusters.end();
		while (iter != iterEnd) {
			float3 difference = point - *iter;
			float distance = sqrt(dot(difference, difference));
			if (distance <= clusterEps) {
				int clusterIndex = iter - clusters.begin();
				clusterPoints[i] = clusterIndex;
				break;
			}
			iter++;
		}
		if (iter == iterEnd) {
			clusters.push_back(point);
			int clusterIndex = clusters.size() - 1;
			clusterPoints[i] = clusterIndex;
		}
	}
	end = std::chrono::high_resolution_clock::now();
	time = std::chrono::duration_cast<std::chrono::duration<float>>(end - start).count();

	/* write to CSV */
	std::ofstream outputFile("output.csv");
	for (int i = 0; i < numPoints; i++) {
		outputFile << h_inputPoints[i] << ",";
		outputFile << h_inputPoints[i + numPoints] << ",";
		outputFile << h_inputPoints[i + 2 * numPoints] << ",";
		outputFile << clusterPoints[i] << "\n";
	}

	printf("\nClustering elapsed time: %f", time);
	printf("\nNum clusters: %lu", clusters.size());
	/* CLUSTERING - fine */

	return 0;
}

